
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel() {
  printf("Hello, world!\n");
}

int main() {
  kernel<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}